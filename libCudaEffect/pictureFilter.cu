#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//

#include "pictureFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "helper_filter.h"
#include <opencv2/core/utility.hpp>
#include <opencv2/core/cuda/common.hpp>
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3      
#define NUM_BINS 256 

using namespace std;


//---------------------------------------------------------------------
//Noise Filter
//---------------------------------------------------------------------

inline  __host__ __device__ float Noise2d(int x, int y)
{
    int n = ((x + (y << 6)) << 13) ^ (x + (y << 6));
    return 0.2f * (1.0f - ((n * (n * n * 15731 + 789221) + 1376312589) & 0x7fffffff) / 1073741824.0);
}

inline  __host__ __device__ float CalculPosValue(int Xint, int Yint)
{
    int m = Xint + ((Yint) << 6);
    int n = (m << 13) ^ (m);
    return (1.0f - ((n * (n * n * 15731 + 789221) + 1376312589) & 0x7fffffff) / 1073741824.0);
}

inline  __host__ __device__ float GetValue(float x, float y)
{
    int Xint = (int)x;
    int Yint = (int)y;

    float Xfrac = x - (float)Xint;
    float Yfrac = y - (float)Yint;

    float x0y0, x1y0, x0y1, x1y1;

    if (Xint != 0 || Yint != 0)
    {
        x0y0 = CalculPosValue(Xint, Yint);
        x1y0 = CalculPosValue(Xint + 1, Yint);
        x0y1 = CalculPosValue(Xint, Yint + 1);
        x1y1 = CalculPosValue(Xint + 1, Yint + 1);
    }
    else
    {
        x0y0 = Noise2d(0, 0);
        x1y0 = Noise2d(1, 0);
        x0y1 = Noise2d(0, 1);
        x1y1 = Noise2d(1, 1);
    }

    //interpolate between those values according to the x and y fractions
    float v1 = (x0y0 + (Xfrac * (x1y0 - x0y0)));
    float v2 = (x0y1 + (Xfrac * (x1y1 - x0y1)));
    float fin = (v1 + (Yfrac * (v2 - v1)));

    return fin;
}

__global__ void noiseFilter(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int position = y * colorWidthStep + (4 * x);
        float4 n = make_float4(Noise2d(x, y));
        float4 src_color = GetColorSrc(x, y, input, colorWidthStep, width, height) + n * 255.0f;
        float4 minimal = make_float4(0.0f);
        float4 maximal = make_float4(255.0f);
        src_color = clamp(src_color, minimal, maximal);
        rgbaFloat4ToUchar4(output, position, src_color, 1.0f);
    }
}

//---------------------------------------------------------------------
//Application du filtre Motion Blur
//---------------------------------------------------------------------
__global__ void motionBlur(uchar* input, uchar* output, int width, int height, const float* kernelMotion, const int2* offsets, int kernelSize, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int position = y * colorWidthStep + (4 * x);
        float4 sum = make_float4(0);


        for (int i = 0; i < kernelSize; i++)
        {
            int u = x + offsets[i].x;
            int v = y + offsets[i].y;
            if ((u < 0) || (u >= width) || (v < 0) || (v >= height))
                continue;
            sum += make_float4(kernelMotion[i])*  GetColorSrc(u, v, input, colorWidthStep, width, height);
        }

        //sum = sum / make_float4(kernelSize);

        rgbaFloat4ToUchar4(output, position, sum, 1.0f);

    }
}

//---------------------------------------------------------------------
//Swirl Filter
//---------------------------------------------------------------------

inline  __host__ __device__ float EuclideanDist(float tcX, float tcY, float centerX, float centerY)
{
    float diffX = tcX - centerX;
    float diffY = tcY - centerY;
    return sqrt(diffX * diffX + diffY * diffY);
}

inline  __host__ __device__ float DotProduct(float tcX, float tcY, float qX, float qY)
{
    return tcX * qX + tcY * qY;
}

inline  __host__ __device__ float4 PostFX(uchar* input, int x, int y, float radius, float angleDegree, int colorWidthStep, int widthIn, int heightIn)
{
    float xOut = x;
    float yOut = y;

    //Calcul du centre
    float centerX = (float)widthIn / 2.0f;
    float centerY = (float)heightIn / 2.0f;

    float tcX = (float)x - centerX;
    float tcY = (float)y - centerY;

    float angle = angleDegree * 0.0174532925;

    float dist = EuclideanDist(x, y, centerX, centerY);

    if (dist < radius)
    {
        float percent = (radius - dist) / radius;
        float theta = percent * percent * angle * 8.0;
        float s = sin(theta);
        float c = cos(theta);
        xOut = DotProduct(tcX, tcY, c, -s);
        yOut = DotProduct(tcX, tcY, s, c);

        tcX = xOut;
        tcY = yOut;
    }

    tcX += centerX;
    tcY += centerY;

    return GetColorSrc((int)tcX, (int)tcY, input, colorWidthStep, widthIn, heightIn);
}

__global__ void swirlFilter(uchar* input, uchar* output, int width, int height, float radius, float angleDegree, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int position = y * colorWidthStep + (4 * x);
        float4 color = PostFX(input, x, y, radius, angleDegree, colorWidthStep, width, height);
        rgbaFloat4ToUchar4(output, position, color, 1.0f);
    }
}

//---------------------------------------------------------------------
//Application du filtre Soften
//	kernel = {  1, 1, 1, 1, 8, 1, 1, 1, 1 };
//	factor = 16;
//---------------------------------------------------------------------
__global__ void softenFilter(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int position = y * colorWidthStep + (4 * x);
        float4 sum = GetColorSrc(x - 1, y - 1, input, colorWidthStep, width, height);
        sum += GetColorSrc(x, y - 1, input, colorWidthStep, width, height);
        sum += GetColorSrc(x + 1, y - 1, input, colorWidthStep, width, height);
        sum += GetColorSrc(x - 1, y, input, colorWidthStep, width, height);
        sum += GetColorSrc(x, y, input, colorWidthStep, width, height) * make_float4(8.0f);
        sum += GetColorSrc(x + 1, y, input, colorWidthStep, width, height);
        sum += GetColorSrc(x - 1, y + 1, input, colorWidthStep, width, height);
        sum += GetColorSrc(x, y + 1, input, colorWidthStep, width, height);
        sum += GetColorSrc(x + 1, y + 1, input, colorWidthStep, width, height);
        sum = sum / make_float4(16.0f);
        rgbaFloat4ToUchar4(output, position, sum, 1.0f);
    }
}


//----------------------------------------------------
//Filtre Posterization
//----------------------------------------------------
__global__ void distorsionFilter(uchar* input, uchar* output, int width, int height, float correctionRadius, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int color_tid = y * colorWidthStep + (4 * x);

        float theta = 1;
        int halfWidth = (width / 2);
        int halfHeight = (height / 2);

        float newX = x - halfWidth;
        float newY = y - halfHeight;
        float value = pow((float)newX, (float)2.0) + pow((float)newY, (float)2.0);
        float distance = sqrt(value);
        float r = distance / correctionRadius;
        if (r != 0.0)
            theta = atan(r) / r;

        int sourceX = round(halfWidth + theta * newX);
        int sourceY = round(halfHeight + theta * newY);

        int positionSrc = sourceY * colorWidthStep + (4 * sourceX);

        ColorRef colorIn = GetfColor(input, positionSrc);

        SetColor(output, color_tid, colorIn);

    }
}
//----------------------------------------------------
//solarization Filter
//----------------------------------------------------
// Run Sharpening Filter on GPU
__global__ void posterisationFilter(uchar* input, uchar* output, unsigned int width, unsigned int height, int level, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int color_tid = y * colorWidthStep + (4 * x);

        //Location of gray pixel in output
        const int gray_tid = y * grayWidthStep + (4 * x);

        ColorRef colorIn = GetfColor(input, color_tid);
        ColorRef colorOut;

        int _levels = max(2, min(16, level));
        float _offset = (float)256 / (float)_levels;

        int red = colorIn.red / _offset;
        int green = colorIn.green / _offset;
        int blue = colorIn.blue / _offset;

        colorOut.red = (red * _offset);
        colorOut.green = (green * _offset);
        colorOut.blue = (blue * _offset);
        colorOut.alpha = colorIn.alpha;

        SetColor(output, gray_tid, colorOut);

    }
}


//----------------------------------------------------
//solarization Filter
//----------------------------------------------------
__global__ void solarizationFilter(uchar* input, uchar* output, unsigned int width, unsigned int height, float threshold, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int color_tid = y * colorWidthStep + (4 * x);
        float4 colorInput = GetColorSrc(x, y, input, colorWidthStep, width, height);
        float4 colorOutput = colorInput;

        float red = colorInput.x;
        float green = colorInput.y;
        float blue = colorInput.z;
        float fthreshold = threshold;

        if (red > fthreshold)
            colorOutput.x = 255.0f - red;
        else
            colorOutput.x = red;

        if (green > fthreshold)
            colorOutput.y = 255.0f - green;
        else
            colorOutput.y = green;

        if (blue > fthreshold)
            colorOutput.z = 255.0f - blue;
        else
            colorOutput.z = blue;

        rgbaFloat4ToUchar4(output, color_tid, colorOutput, 1.0f);
    }
}

//----------------------------------------------------
//dilate Filter
//----------------------------------------------------

__global__ void erodeFilter(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int color_tid = y * colorWidthStep + (4 * x);
        float4 minValue = make_float4(255.0f);

        for (int n = -1; n < 2; n++)
        {
            int posY = y + n;
            for (int m = -1; m < 2; m++)
            {
                int posX = x + m;
                if ((posX < width) && (posY < height))
                    minValue = fminf(GetColorSrc(posX, posY, input, colorWidthStep, width, height), minValue);
            }
        }

        rgbaFloat4ToUchar4(output, color_tid, minValue, 1.0f);
    }
}

__global__ void dilateFilter(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        const int color_tid = y * colorWidthStep + (4 * x);
        float4 maxValue = make_float4(0.0f);

        for (int n = -1; n < 2; n++)
        {
            int posY = y + n;
            for (int m = -1; m < 2; m++)
            {
                int posX = x + m;
                if ((posX < width) && (posY < height))
                    maxValue = fmaxf(GetColorSrc(posX, posY, input, colorWidthStep, width, height), maxValue);
            }
        }

        rgbaFloat4ToUchar4(output, color_tid, maxValue, 1.0f);
    }
}

//----------------------------------------------------
//sharpenMasking Filter
//----------------------------------------------------

__global__ void sharpenMasking(uchar* input, uchar* output, uchar* gaussian, int width, int height, float sharpness, int colorWidthStep, int grayWidthStep)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        //Location of gray pixel in output
        const int gray_tid = yIndex * grayWidthStep + (4 * xIndex);

        ColorRef colorIn = GetfColor(input, color_tid);
        ColorRef colorGaussian = GetfColor(gaussian, color_tid);

        ColorRef color = colorIn - colorGaussian;

        ColorRef value = colorIn + color * sharpness;

        value = clamp(value, 0.0f, 255.0f);

        SetColor(output, gray_tid, value);
    }
}

//----------------------------------------------------
//Filtre Sepia
//----------------------------------------------------
__global__ void sepiaFilter(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        //Location of gray pixel in output
        const int gray_tid = yIndex * grayWidthStep + (4 * xIndex);

        ColorRef color = GetfColor(input, color_tid);
        ColorRef colorOut;

        colorOut.red = (0.393f * color.blue) + (0.769f * color.green) + (0.189f * color.red);
        colorOut.green = (0.349f * color.blue) + (0.686f * color.green) + (0.168f * color.red);
        colorOut.blue = (0.272f * color.blue) + (0.534f * color.green) + (0.131f * color.red);

        color = clamp(colorOut, 0.0, 255.0);
        
        SetColor(output, gray_tid, color);

    }
}

//----------------------------------------------------
//Filtre Mosaic
//----------------------------------------------------
__global__ void mosaicFilter(uchar* input, uchar* output, int width, int height, int fTileSize, int colorWidthStep, int grayWidthStep)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        float w = fTileSize;
        float h = fTileSize;

        float s = floor(xIndex / w);
        float t = floor(yIndex / h);

        int sourcex = __float2int_rd(s * w);
        int sourcey = __float2int_rd(t * h);

        int positionSrc = sourcey * colorWidthStep + (4 * sourcex);

        ColorRef colorIn = GetfColor(input, positionSrc);

        SetColor(output, color_tid, colorIn);

    }
}

//----------------------------------------------------
//Filtre Median
//----------------------------------------------------


inline  __host__ __device__ void s2(float4* a, float4* b)
{
    float4 temp = *a;
    *a = fminf(*a, *b);
    *b = fmaxf(temp, *b);
}

inline  __host__ __device__ void mn3(float4* a, float4* b, float4* c)
{
    s2(a, b);
    s2(a, c);
}

inline  __host__ __device__ void mx3(float4* a, float4* b, float4* c)
{
    s2(b, c);
    s2(a, c);
}

inline  __host__ __device__ void mnmx3(float4* a, float4* b, float4* c)
{
    mx3(a, b, c);
    s2(a, b);
}

inline  __host__ __device__ void mnmx4(float4* a, float4* b, float4* c, float4* d)
{
    s2(a, b); s2(c, d); s2(a, c); s2(b, d);
}

inline  __host__ __device__ void mnmx5(float4* a, float4* b, float4* c, float4* d, float4* e)
{
    s2(a, b); s2(c, d); mn3(a, c, e); mx3(b, d, e);
}

inline  __host__ __device__ void mnmx6(float4* a, float4* b, float4* c, float4* d, float4* e, float4* f)
{
    s2(a, d); s2(b, e); s2(c, f); mn3(a, b, c); mx3(d, e, f);
}

__global__ void medianFilter(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((x < width) && (y < height))
    {
        //Location of colored pixel in input
        const int color_tid = y * colorWidthStep + (4 * x);
        float4 k0 = GetColorSrc(x - 1, y - 1, input, colorWidthStep, width, height);
        float4 k1 = GetColorSrc(x, y - 1, input, colorWidthStep, width, height);
        float4 k2 = GetColorSrc(x + 1, y - 1, input, colorWidthStep, width, height);
        float4 k3 = GetColorSrc(x - 1, y, input, colorWidthStep, width, height);
        float4 k4 = GetColorSrc(x, y, input, colorWidthStep, width, height);
        float4 k5 = GetColorSrc(x + 1, y, input, colorWidthStep, width, height);
        float4 k6 = GetColorSrc(x - 1, y + 1, input, colorWidthStep, width, height);
        float4 k7 = GetColorSrc(x, y + 1, input, colorWidthStep, width, height);
        float4 k8 = GetColorSrc(x + 1, y + 1, input, colorWidthStep, width, height);

        mnmx6(&k0, &k1, &k2, &k3, &k4, &k5);
        mnmx5(&k1, &k2, &k3, &k4, &k6);
        mnmx4(&k2, &k3, &k4, &k7);
        mnmx3(&k3, &k4, &k8);

        rgbaFloat4ToUchar4(output, color_tid, k4, 1.0f);
    }

}

//----------------------------------------------------
//Filtre Photo Filtre
//----------------------------------------------------
__global__  void photoFilter(uchar* input, uchar* output, int width, int height, float intensity, uint4 color, int colorWidthStep, int grayWidthStep)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        //Location of gray pixel in output
        const int gray_tid = yIndex * grayWidthStep + (4 * xIndex);

        ColorRef colorIn = GetfColor(input, color_tid);
        ColorRef colorOut;

        float coeff = intensity / 100.0f;
        float diff = 1.0f - coeff;

        colorOut.red = (float)(color.x) * coeff + colorIn.red * diff;
        colorOut.green = (float)(color.y) * coeff + colorIn.green * diff;
        colorOut.blue = (float)(color.z) * coeff + colorIn.blue * diff;

        colorIn = clamp(colorOut, 0.0, 255.0);

        SetColor(output, gray_tid, colorIn);
    }

}


__global__ void histogram(const uchar *in, int width, int height, int colorWidthStep, unsigned int *out)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // initialize temporary accumulation array in shared memory
   // __shared__ unsigned int smem[NUM_BINS];
   // for (int i = 0; i < NUM_BINS; i++) 
    //    smem[i] = 0;
    __syncthreads();
    
    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        const int gray_tid = yIndex * colorWidthStep + (xIndex);
        unsigned int r = (unsigned int)(in[gray_tid]);
        atomicAdd(&out[r], 1);
    }
    __syncthreads();

    //for (int i = 0; i < NUM_BINS; i++) {
    //    out[i] = smem[i];
    //}
}

//----------------------------------------------------
//Filtre Auto Contrast
//----------------------------------------------------



__global__ void autoConstrast(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep, float min, float  max)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        float constant = 255.0f / (max - min);
        ColorRef colorIn = GetfColor(input, color_tid);
        colorIn = (colorIn - (min)) * (constant);
        colorIn = clamp(colorIn, 0.0, 255.0);
        SetColor(output, color_tid, colorIn);

    }
}

__global__ void convertScaleAbs(uchar* input, uchar* output, int width, int height, int colorWidthStep, int grayWidthStep, float alpha, float  beta)
{
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {

        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        //Location of gray pixel in output
        const int gray_tid = yIndex * grayWidthStep + (4 * xIndex);

        ColorRef colorIn = GetfColor(input, color_tid);
        colorIn = abs(colorIn * alpha + beta);
        colorIn = clamp(colorIn, 0.0, 255.0);
        SetColor(output, gray_tid, colorIn);
        /*
        float4 color = GetfloatColor(input, color_tid);
        color.x = fabs(color.x * alpha + beta);
        color.y = fabs(color.y * alpha + beta);
        color.w = fabs(color.w * alpha + beta);
        color.z = fabs(color.z * alpha + beta);
        
        rgbaFloat4ToUchar4(output, gray_tid, color, 1.0f);
        */
    }
}


unsigned int * cuda_histogram(const cv::cuda::GpuMat& input)
{
    unsigned char* d_input;
    unsigned int * d_output;

    unsigned int* tab = new unsigned int[NUM_BINS];

    d_input = (uchar*)input.ptr();
    
    hipMalloc<unsigned int>(&d_output, NUM_BINS * sizeof(unsigned int));
    
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    histogram << <grid, block >> > (d_input, input.cols, input.rows, input.step, d_output);

    // Synchronize to check for any kernel launch errors
#ifdef WIN32
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
#else
    cudaSafeCall(hipDeviceSynchronize());
#endif 

    hipMemcpy(tab, d_output, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(d_output);
    
    return tab;
}

void cuda_autoContrast(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const float& min, const float& max)
{
    unsigned char* d_input, * d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    autoConstrast << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step, min, max);

    // Synchronize to check for any kernel launch errors
#ifdef WIN32
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
#else
    cudaSafeCall(hipDeviceSynchronize());
#endif 
}

void cuda_convertScaleAbs(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const float & alpha, const float &  beta)
{
    unsigned char* d_input, * d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();
    
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    convertScaleAbs << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step, alpha, beta);

    // Synchronize to check for any kernel launch errors
#ifdef WIN32
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
#else
    cudaSafeCall(hipDeviceSynchronize());
#endif 
}

void CCudaComputeFilter::ApplyEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    ExecuteEffect(input, output);

    // Synchronize to check for any kernel launch errors
#ifdef WIN32
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
#else
    cudaSafeCall(hipDeviceSynchronize());
#endif 
}

void CSepiaFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    sepiaFilter << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);

}

void CPhotoFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    uint4 color;
    color.x = clValue.GetRed();
    color.y = clValue.GetGreen();
    color.z = clValue.GetBlue();

    // Launch the color conversion kernel
    photoFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, intensity, color, input.step, output.step);
}

void CMosaicFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    mosaicFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, fTileSize, input.step, output.step);
}

// The wrapper is used to call sharpening filter 
void CCudaSharpenMaskingFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    sharpenMasking << <grid, block >> > (d_input, d_output, gaussian_input, output.cols, output.rows, sharpness, input.step, output.step);

}

void CCudaMedianFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    medianFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, input.step, output.step);
}

void CDilateFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    dilateFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, input.step, output.step);
}

void CErodeFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    erodeFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, input.step, output.step);
}


// The wrapper is used to call sharpening filter 
void CSolarizationFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    solarizationFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, level, input.step, output.step);
}

// The wrapper is used to call sharpening filter 
void CPosterizationFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    posterisationFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, level, input.step, output.step);
}

// The wrapper is used to call sharpening filter 
void CDistorsionFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    distorsionFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, correctionRadius, input.step, output.step);
}

void CSoftenFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    softenFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, input.step, output.step);
}


void CSwirlFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    swirlFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, radius, angleDegree, input.step, output.step);
}

void CNoiseFilter::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    noiseFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, input.step, output.step);
}

void CMotionBlur::SetParameter(const vector<double>& kernelMotion, const vector<Points>& offsets, int kernelSize)
{
    offsetSize = offsets.size();
    kSize = kernelMotion.size();
    kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    offsetsMotion = new int2[offsets.size()];
    for (auto i = 0; i < offsets.size(); i++)
    {
        offsetsMotion[i].x = offsets[i].x;
        offsetsMotion[i].y = offsets[i].y;
    }
    this->kernelSize = kernelSize;
}

void CMotionBlur::ExecuteEffect(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    float* f_kernel;
    int2* i_offsetsMotion;
    // Allocate device memory
    hipMalloc<float>(&f_kernel, kSize * sizeof(float));
    hipMalloc<int2>(&i_offsetsMotion, offsetSize * sizeof(int2));

    hipMemcpy(f_kernel, kernel, kSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(i_offsetsMotion, offsetsMotion, offsetSize * sizeof(int2), hipMemcpyHostToDevice);

    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    motionBlur << <grid, block >> > (d_input, d_output, input.cols, input.rows, f_kernel, i_offsetsMotion, kernelSize, input.step, output.step);

    // Synchronize to check for any kernel launch errors
#ifdef WIN32
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
#else
    cudaSafeCall(hipDeviceSynchronize());
#endif 
    hipFree(f_kernel);
    hipFree(offsetsMotion);

}


