#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//

#include "pictureFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

inline  __host__ __device__  uchar4 GetColorSrc(unsigned char* input, int position)
{
    uchar4 value;
    value.x = input[position];
    value.y = input[position + 1];
    value.z = input[position + 2];
    value.w = input[position + 3];
    return value;
}

//----------------------------------------------------
// Conversion du NV12 vers du 32 bits
//----------------------------------------------------
inline  __host__ __device__ void rgbaFloat4ToUchar4(unsigned char*& output, int position, float4 rgba, float fScale)
{
    output[position] = (uchar)(rgba.x * fScale);
    output[position + 1] = (uchar)(rgba.y * fScale);
    output[position + 2] = (uchar)(rgba.z * fScale);
    output[position + 3] = (uchar)(rgba.w * fScale);

}


inline  __host__ __device__  float4 GetColorSrc(int x, int y, const unsigned char* input, int width, int height)
{
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        int position = (x + y * width) * 4;
        float4 value;
        value.x = (float)input[position];
        value.y = (float)input[position + 1];
        value.z = (float)input[position + 2];
        value.w = (float)input[position + 3];
        return value;
    }
    return make_float4(0.0f, 0.0f, 0.0f,0.0f);
}


//----------------------------------------------------
//Filtre Niveau de gris
//----------------------------------------------------


// Run Sharpening Filter on GPU
__global__ void mosaicFilter(uchar*srcImage, uchar*dstImage, unsigned int width, unsigned int height, int fTileSize)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float w = fTileSize;
	    float h = fTileSize;		

	    float s = floor(x / w);
	    float t = floor(y / h);	
	    
	    int sourcex = __float2int_rd(s * w);
	    int sourcey = __float2int_rd(t * h);
	    
	    int positionSrc = (sourcex + sourcey * width) * 4;
	    int positionDest = (x + y * width) * 4;

        dstImage[positionDest] = srcImage[positionSrc];
        dstImage[positionDest + 1] = srcImage[positionSrc + 1];
        dstImage[positionDest + 2] = srcImage[positionSrc + 2];
        dstImage[positionDest + 3] = srcImage[positionSrc + 3];
    }
}


// Run Sharpening Filter on GPU
__global__ void solarizationFilter(uchar*srcImage, uchar*dstImage, unsigned int width, unsigned int height, int threshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float4 colorInput = GetColorSrc(x,y,srcImage,width,height);
	    float4 colorOutput = colorInput;
	    
	    float red = colorInput.x;
	    float green = colorInput.y;
	    float blue = colorInput.z;
	    float fthreshold = (float)threshold;
	    
	    if (red > fthreshold)
		    colorOutput.x = 255.0f - red;
	    else
		    colorOutput.x = red;
		    
	    if (green > fthreshold)
		    colorOutput.y = 255.0f - green;
	    else
		    colorOutput.y = green;

	    if (blue > fthreshold)
		    colorOutput.z = 255.0f - blue;
	    else
		    colorOutput.z = blue;

        dstImage[position] = colorOutput.x;
        dstImage[position + 1] = colorOutput.y;
        dstImage[position + 2] = colorOutput.z;
        dstImage[position + 3] = colorOutput.w;
    }
}


// Run Sharpening Filter on GPU
__global__ void posterisationFilter(uchar* srcImage, uchar* dstImage, unsigned int width, unsigned int height, int level)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = (x + y * width) * 4;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    uchar4 colorInput = GetColorSrc(srcImage,position);
	    uchar4 colorOutput = colorInput;
	    int _levels = max(2, min(16, level));
	    float _offset = (float)256 / (float)_levels;
	    
	    int red = colorInput.x / _offset;
	    int green = colorInput.y / _offset;
	    int blue = colorInput.z / _offset;
	    
	    colorOutput.x = (red * _offset);
	    colorOutput.y = (green * _offset);
	    colorOutput.z = (blue * _offset);

        dstImage[position] = colorOutput.x;
        dstImage[position+1] = colorOutput.y;
        dstImage[position+2] = colorOutput.z;
        dstImage[position+3] = colorOutput.w;
    }
}



//----------------------------------------------------
//Filtre Posterization
//----------------------------------------------------
__global__ void distorsionFilter(uchar* input, uchar* output,  int width, int height, float correctionRadius)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = (x + y * width) * 4;

    if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float theta = 1;
	    int halfWidth = (width / 2); 
	    int halfHeight = (height / 2);
	    
	    float newX = x - halfWidth; 
	    float newY = y - halfHeight;
	    float value = pow((float)newX,(float)2.0) + pow((float)newY, (float)2.0);
	    float distance = sqrt(value);
	    float r = distance / correctionRadius;
	    if (r != 0.0)
		    theta = atan(r) / r;
		    
	    int sourceX = round(halfWidth + theta*newX);
	    int sourceY = round(halfHeight + theta * newY);
	    
	    int positionSrc = (sourceX + sourceY * width) * 4;

	    output[position] = input[positionSrc];
        output[position+1] = input[positionSrc + 1];
        output[position + 2] = input[positionSrc + 2];
        output[position + 3] = input[positionSrc + 3];
    }
}

//---------------------------------------------------------------------
//Limite les valeurs entre 0 et 1.0f
//---------------------------------------------------------------------
inline  __host__ __device__ float4 NormalizeValue(float4 sum)
{
    float4 value;
    value.x = max(min(sum.x, 255.0f), 0.0f);
    value.y = max(min(sum.y, 255.0f), 0.0f);
    value.z = max(min(sum.z, 255.0f), 0.0f);
    value.w = max(min(sum.w, 255.0f), 0.0f);
    return value;
}

//---------------------------------------------------------------------
//Application du filtre Motion Blur
//---------------------------------------------------------------------
__global__ void motionBlur(uchar* input, uchar * output,  int width, int height, const float* kernelMotion, const int2 * offsets, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 sum = make_float4(0);
        for (int i = 0; i < kernelSize; i++)
        {
            int u = x + offsets[i].x;
            int v = y + offsets[i].y;
            if ((u < 0) || (u >= width) || (v < 0) || (v >= height))
                continue;

            float4 color = kernelMotion[i] * GetColorSrc(u, v, input, width, height);
            sum = sum + color;
        }

        rgbaFloat4ToUchar4(output, position, NormalizeValue(sum), 1.0f);

    }
}

//---------------------------------------------------------------------
//Application du filtre Soften
//	kernel = {  1, 1, 1, 1, 8, 1, 1, 1, 1 };
//	factor = 16;
//---------------------------------------------------------------------
__global__ void softenFilter(uchar* input, uchar * output,  int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 sum = GetColorSrc(x - 1, y - 1, input, width, height);
        sum += GetColorSrc(x, y - 1, input, width, height);
        sum += GetColorSrc(x + 1, y - 1, input, width, height);
        sum += GetColorSrc(x - 1, y, input, width, height);
        sum += GetColorSrc(x, y, input, width, height) * make_float4(8.0f);
        sum += GetColorSrc(x + 1, y, input, width, height);
        sum += GetColorSrc(x - 1, y + 1, input, width, height);
        sum += GetColorSrc(x, y + 1, input, width, height);
        sum += GetColorSrc(x + 1, y + 1, input, width, height);
        sum = sum / make_float4(16.0f);
        rgbaFloat4ToUchar4(output, position, sum, 1.0f);
    }
}

__global__ void sepiaFilter(uchar* input, uchar* output,  int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 color = GetColorSrc(x, y, input, width, height);
        float4 color_out = make_float4(0.0f);
        color_out.x = 0.272 * color.x +0.534 * color.y + 0.131 * color.z;
        color_out.y = 0.349 * color.x + 0.686 * color.y + 0.168 * color.z; //0.349, 0.686, 0.168 
        color_out.z = 0.393 * color.x + 0.769 * color.y + 0.189 * color.z; //0.393, 0.769, 0.189
        color_out.w = color.w;
        rgbaFloat4ToUchar4(output, position, color_out, 1.0f);
    }
}

__global__ void cuda_filter2d(uchar* input, uchar* output, float* kernelMotion, int width, int height, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 data = GetColorSrc(x, y, input, width, height);
        float4 sum = make_float4(0);
        for (int i = 0; i < kernelSize; i++)
        {
            float4 color = kernelMotion[i] * data;
            sum = sum + color;
        }
        rgbaFloat4ToUchar4(output, position, NormalizeValue(sum), 1.0f);
    }
}

__global__ void sharpenMasking(uchar* input, uchar* output, uchar * gaussian, int width, int height, float sharpness)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {

        float4 origin = GetColorSrc(x, y, input, width, height);
        float4 color = GetColorSrc(x, y, gaussian, width, height);

        color = origin - color;

        float4 value = origin + color * sharpness;


        value = clamp(value, 255.0f, 0.0f);

        rgbaFloat4ToUchar4(output, position, value, 1.0f);
    }
}

inline  __host__ __device__ float Noise2d(int x, int y)
{
    int n = ((x + (y << 6)) << 13) ^ (x + (y << 6));
    return 0.2f * (1.0f - ((n * (n * n * 15731 + 789221) + 1376312589) & 0x7fffffff) / 1073741824.0);
}

inline  __host__ __device__ float CalculPosValue(int Xint, int Yint)
{
    int m = Xint + ((Yint) << 6);
    int n = (m << 13) ^ (m);
    return (1.0f - ((n * (n * n * 15731 + 789221) + 1376312589) & 0x7fffffff) / 1073741824.0);
}

inline  __host__ __device__ float GetValue(float x, float y)
{
    int Xint = (int)x;
    int Yint = (int)y;

    float Xfrac = x - (float)Xint;
    float Yfrac = y - (float)Yint;

    float x0y0, x1y0, x0y1, x1y1;

    if (Xint != 0 || Yint != 0)
    {
        x0y0 = CalculPosValue(Xint, Yint);
        x1y0 = CalculPosValue(Xint + 1, Yint);
        x0y1 = CalculPosValue(Xint, Yint + 1);
        x1y1 = CalculPosValue(Xint + 1, Yint + 1);
    }
    else
    {
        x0y0 = Noise2d(0, 0);
        x1y0 = Noise2d(1, 0);
        x0y1 = Noise2d(0, 1);
        x1y1 = Noise2d(1, 1);
    }

    //interpolate between those values according to the x and y fractions
    float v1 = (x0y0 + (Xfrac * (x1y0 - x0y0)));
    float v2 = (x0y1 + (Xfrac * (x1y1 - x0y1)));
    float fin = (v1 + (Yfrac * (v2 - v1)));

    return fin;
}

__global__ void noiseFilter(uchar* input, uchar * output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 n = make_float4(Noise2d(x, y));
        float4 src_color = GetColorSrc(x, y, input, width, height) + n * 255.0f;
        float4 minimal = make_float4(0.0f);
        float4 maximal = make_float4(255.0f);
        src_color = clamp(src_color, minimal, maximal);
        rgbaFloat4ToUchar4(output, position, src_color, 1.0f);
    }
}


inline  __host__ __device__ float EuclideanDist(float tcX, float tcY, float centerX, float centerY)
{
    float diffX = tcX - centerX;
    float diffY = tcY - centerY;
    return sqrt(diffX * diffX + diffY * diffY);
}

inline  __host__ __device__ float DotProduct(float tcX, float tcY, float qX, float qY)
{
    return tcX * qX + tcY * qY;
}

inline  __host__ __device__ float4 PostFX(uchar * input, int x, int y, float radius, float angleDegree, int widthIn, int heightIn)
{
    float xOut = x;
    float yOut = y;

    //Calcul du centre
    float centerX = (float)widthIn / 2.0f;
    float centerY = (float)heightIn / 2.0f;

    float tcX = (float)x - centerX;
    float tcY = (float)y - centerY;

    float angle = angleDegree * 0.0174532925;

    float dist = EuclideanDist(x, y, centerX, centerY);

    if (dist < radius)
    {
        float percent = (radius - dist) / radius;
        float theta = percent * percent * angle * 8.0;
        float s = sin(theta);
        float c = cos(theta);
        xOut = DotProduct(tcX, tcY, c, -s);
        yOut = DotProduct(tcX, tcY, s, c);

        tcX = xOut;
        tcY = yOut;
    }

    tcX += centerX;
    tcY += centerY;

    return GetColorSrc((int)tcX, (int)tcY, input, widthIn, heightIn);
}

__global__ void swirlFilter(uchar* input, uchar * output,  int width, int height, float radius, float angleDegree)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 color = PostFX(input, x, y, radius, angleDegree, width, height);
        rgbaFloat4ToUchar4(output, position, color, 1.0f);
    }
}

__global__ void erodeFilter(uchar* input, uchar* output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 maxValue = make_float4(0.0f);

        for (int n = -1; n < 2; n++)
        {
            int posY = y + n;
            for (int m = -1; m < 2; m++)
            {
                int posX = x + m;
                maxValue = fmaxf(GetColorSrc(posX, posY, input, width, height), maxValue);
            }
        }

        rgbaFloat4ToUchar4(output, position, maxValue, 1.0f);
    }
}

__global__ void dilateFilter(uchar* input, uchar* output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 minValue = make_float4(1.0f);

        for (int n = -1; n < 2; n++)
        {
            int posY = y + n;
            for (int m = -1; m < 2; m++)
            {
                int posX = x + m;
                minValue = fminf(GetColorSrc(posX, posY, input, width, height), minValue);
            }
        }

        rgbaFloat4ToUchar4(output, position, minValue, 1.0f);
    }
}


inline  __host__ __device__ void s2(float4* a, float4* b)
{
    float4 temp = *a;
    *a = fminf(*a, *b);
    *b = fmaxf(temp, *b);
}

inline  __host__ __device__ void mn3(float4* a, float4* b, float4* c)
{
    s2(a, b);
    s2(a, c);
}

inline  __host__ __device__ void mx3(float4* a, float4* b, float4* c)
{
    s2(b, c);
    s2(a, c);
}

inline  __host__ __device__ void mnmx3(float4* a, float4* b, float4* c)
{
    mx3(a, b, c);
    s2(a, b);
}

inline  __host__ __device__ void mnmx4(float4* a, float4* b, float4* c, float4* d)
{
    s2(a, b); s2(c, d); s2(a, c); s2(b, d);
}

inline  __host__ __device__ void mnmx5(float4* a, float4* b, float4* c, float4* d, float4* e)
{
    s2(a, b); s2(c, d); mn3(a, c, e); mx3(b, d, e);
}

inline  __host__ __device__ void mnmx6(float4* a, float4* b, float4* c, float4* d, float4* e, float4* f)
{
    s2(a, d); s2(b, e); s2(c, f); mn3(a, b, c); mx3(d, e, f);
}

__global__ void medianFilter(uchar* input, uchar* output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 k0 = GetColorSrc(x - 1, y - 1, input, width, height);
        float4 k1 = GetColorSrc(x, y - 1, input, width, height);
        float4 k2 = GetColorSrc(x + 1, y - 1, input, width, height);
        float4 k3 = GetColorSrc(x - 1, y, input, width, height);
        float4 k4 = GetColorSrc(x, y, input, width, height);
        float4 k5 = GetColorSrc(x + 1, y, input, width, height);
        float4 k6 = GetColorSrc(x - 1, y + 1, input, width, height);
        float4 k7 = GetColorSrc(x, y + 1, input, width, height);
        float4 k8 = GetColorSrc(x + 1, y + 1, input, width, height);

        mnmx6(&k0, &k1, &k2, &k3, &k4, &k5);
        mnmx5(&k1, &k2, &k3, &k4, &k6);
        mnmx4(&k2, &k3, &k4, &k7);
        mnmx3(&k3, &k4, &k8);

        rgbaFloat4ToUchar4(output, position, k4, 1.0f);
    }

}

void cuda_filter2d(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const vector<float>& kernelMotion, int kernelSize)
{
    uchar* d_input;
    uchar* d_output;
    float* f_kernel;

    float* kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    hipMalloc<float>(&f_kernel, kernelMotion.size());

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    cuda_filter2d << <grid, block >> > (d_input, d_output, f_kernel, output.cols, output.rows, kernelSize);

    hipFree(f_kernel);
    delete[] kernel;
}

// The wrapper is used to call sharpening filter 
void medianFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    medianFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}


// The wrapper is used to call sharpening filter 
void dilateFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    dilateFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}

// The wrapper is used to call sharpening filter 
void erodeFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    erodeFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}

// The wrapper is used to call sharpening filter 
void noiseFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    noiseFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}


// The wrapper is used to call sharpening filter 
void swirlFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, float radius, float angleDegree)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    swirlFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, radius, angleDegree);
}


// The wrapper is used to call sharpening filter 
void sepiaFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    sepiaFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}

// The wrapper is used to call sharpening filter 
void softenFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    softenFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}

// The wrapper is used to call sharpening filter 
void distorsionFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, float correctionRadius)
{
    uchar * d_input;
    uchar * d_output;

    d_input = (uchar * )input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

    // Run BoxFilter kernel on CUDA 
    distorsionFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, correctionRadius);
}



// The wrapper is used to call sharpening filter 
void posterisationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int threshold)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    posterisationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, threshold);
}


void motionBlur(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const vector<double>& kernelMotion, const vector<wxPoint>& offsets, int kernelSize)
{
    uchar* d_input;
    uchar* d_output;
    float* f_kernel;
    int2 * i_offsetsMotion;

    float * kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    int2* offsetsMotion = new int2[offsets.size()];
    for (auto i = 0; i < offsets.size(); i++)
    {
        offsetsMotion[i].x = offsets[i].x;
        offsetsMotion[i].y = offsets[i].y;
    }

    // Allocate device memory
    hipMalloc<float>(&f_kernel, kernelMotion.size());
    hipMalloc<int2>(&i_offsetsMotion, offsets.size());

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    hipMemcpy(f_kernel, kernel, kernelMotion.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(i_offsetsMotion, offsetsMotion, offsets.size() * sizeof(int2), hipMemcpyHostToDevice);

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    motionBlur << <grid, block >> > (d_input, d_output, output.cols, output.rows, f_kernel, i_offsetsMotion, kernelSize);

    hipFree(f_kernel);
    hipFree(offsetsMotion);

    delete[] i_offsetsMotion;
    delete[] f_kernel;
}


// The wrapper is used to call sharpening filter 
void solarizationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int level)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    solarizationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, level);
}


// The wrapper is used to call sharpening filter 
void mosaicFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int fTileSize)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    mosaicFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, fTileSize);

}

// The wrapper is used to call sharpening filter 
void sharpenMasking(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const cv::cuda::GpuMat& gaussian, float sharpness)
{
    uchar* d_input;
    uchar* d_output;
    uchar* gaussian_input;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();
    gaussian_input = (uchar*)gaussian.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    sharpenMasking << <grid, block >> > (d_input, d_output, gaussian_input, output.cols, output.rows, sharpness);

}










