#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//

#include "pictureFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include <opencv2/core/utility.hpp>
#include <opencv2/core/cuda/common.hpp>
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

 __host__ __device__ struct ColorRef
{
     uchar red;
     uchar green;
     uchar blue;
     uchar alpha;
};

inline  __host__ __device__  uchar4 GetColorSrc(unsigned char* input, int position)
{
    uchar4 value;
    value.x = input[position];
    value.y = input[position + 1];
    value.z = input[position + 2];
    value.w = input[position + 3];
    return value;
}

inline  __host__ __device__  float4 GetfColorSrc(unsigned char* input, int position)
{
    float4 value;
    value.x = input[position];
    value.y = input[position + 1];
    value.z = input[position + 2];
    value.w = input[position + 3];
    return value;
}

//----------------------------------------------------
// Conversion du NV12 vers du 32 bits
//----------------------------------------------------
inline  __host__ __device__ void rgbaFloat4ToUchar4(uchar * output, int position, float4 rgba, float fScale)
{
    output[position] = (uchar)(rgba.x * fScale);
    output[position + 1] = (uchar)(rgba.y * fScale);
    output[position + 2] = (uchar)(rgba.z * fScale);
    output[position + 3] = (uchar)(rgba.w * fScale);
    output[position] = clamp(output[position], 0, 255);
    output[position + 1] = clamp(output[position + 1], 0, 255);
    output[position + 2] = clamp(output[position + 2], 0, 255);
    output[position + 3] = clamp(output[position + 3], 0, 255);
}


inline  __host__ __device__  float4 GetColorSrc(int x, int y, uchar * input, int width, int height)
{
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        int position = (x + y * width) * 4;
        float4 value;
        value.x = (float)input[position];
        value.y = (float)input[position + 1];
        value.z = (float)input[position + 2];
        value.w = (float)input[position + 3];
        return value;
    }
    return make_float4(0.0f, 0.0f, 0.0f,0.0f);
}


//----------------------------------------------------
//Filtre Niveau de gris
//----------------------------------------------------



//---------------------------------------------------------------------
//Limite les valeurs entre 0 et 1.0f
//---------------------------------------------------------------------
inline  __host__ __device__ float4 NormalizeValue(float4 sum)
{
    float4 value;
    value.x = max(min(sum.x, 255.0f), 0.0f);
    value.y = max(min(sum.y, 255.0f), 0.0f);
    value.z = max(min(sum.z, 255.0f), 0.0f);
    value.w = max(min(sum.w, 255.0f), 0.0f);
    return value;
}

//---------------------------------------------------------------------
//Application du filtre Motion Blur
//---------------------------------------------------------------------
__global__ void motionBlur(uchar* input, uchar * output,  int width, int height, const float* kernelMotion, const int2 * offsets, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 sum = make_float4(0);
        for (int i = 0; i < kernelSize; i++)
        {
            int u = x + offsets[i].x;
            int v = y + offsets[i].y;
            if ((u < 0) || (u >= width) || (v < 0) || (v >= height))
                continue;

            float4 color = kernelMotion[i] * GetColorSrc(u, v, input, width, height);
            sum = sum + color;
        }

        rgbaFloat4ToUchar4(output, position, NormalizeValue(sum), 1.0f);

    }
}





inline __host__ __device__ float4 make_float4(uchar4 value)
{
    float4 color = make_float4(0.0);
    color.x = value.x;
    color.y = value.y;
    color.z = value.z;
    color.w = value.w;
    return color;
}

inline __host__ __device__ uchar4 make_uchar4(float4 value)
{
    uchar4 color;;
    color.x = clamp((uchar)value.x,0,255);
    color.y = clamp((uchar)value.y, 0, 255);
    color.z = clamp((uchar)value.z, 0, 255);
    color.w = clamp((uchar)value.w, 0, 255);
    return color;
}

__global__ void cuda_filter2d(uchar* input, uchar* output, float* kernelMotion, int width, int height, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 data = GetColorSrc(x, y, input, width, height);
        float4 sum = make_float4(0);
        for (int i = 0; i < kernelSize; i++)
        {
            float4 color = kernelMotion[i] * data;
            sum = sum + color;
        }
        rgbaFloat4ToUchar4(output, position, NormalizeValue(sum), 1.0f);
    }
}



inline  __host__ __device__ ColorRef GetColor(uchar* input, const int color_tid)
{
    ColorRef color;
    color.blue = input[color_tid];
    color.green = input[color_tid + 1];
    color.red = input[color_tid + 2];
    color.alpha = input[color_tid + 3];
    return color;
}

/**
 * @brief      BGR to Gray Kernel
 *
 *             This is a simple image processing kernel that converts color
 *             images to black and white by iterating over the individual
 *             pixels.
 *
 * @param      input           The input
 * @param      output          The output
 * @param[in]  width           The width
 * @param[in]  height          The height
 * @param[in]  colorWidthStep  The color width step
 * @param[in]  grayWidthStep   The gray width step
 */
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep) {
    //2D Index of current thread
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        //Location of gray pixel in output
        const int gray_tid = yIndex * grayWidthStep + (4 * xIndex);

        const unsigned char blue = input[color_tid];
        const unsigned char green = input[color_tid + 1];
        const unsigned char red = input[color_tid + 2];

        const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

        output[gray_tid] = static_cast<unsigned char>(gray);
        output[gray_tid + 1] = static_cast<unsigned char>(gray);
        output[gray_tid + 2] = static_cast<unsigned char>(gray);
        output[gray_tid + 3] = static_cast<unsigned char>(gray);
    }
}


void convert_to_gray(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output) {
    // Calculate total number of bytes of input and output image
    const int colorBytes = input.step * input.rows;
    const int grayBytes = output.step * output.rows;

    unsigned char* d_input, * d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    bgr_to_gray_kernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);

    // Synchronize to check for any kernel launch errors
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
}


void cuda_filter2d(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const vector<float>& kernelMotion, int kernelSize)
{
    uchar* d_input;
    uchar* d_output;
    float* f_kernel;

    float* kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    hipMalloc<float>(&f_kernel, kernelMotion.size());

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    cuda_filter2d << <grid, block >> > (d_input, d_output, f_kernel, output.cols, output.rows, kernelSize);

    hipFree(f_kernel);
    delete[] kernel;
}




void motionBlur(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const vector<double>& kernelMotion, const vector<wxPoint>& offsets, int kernelSize)
{
    uchar* d_input;
    uchar* d_output;
    float* f_kernel;
    int2 * i_offsetsMotion;

    float * kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    int2* offsetsMotion = new int2[offsets.size()];
    for (auto i = 0; i < offsets.size(); i++)
    {
        offsetsMotion[i].x = offsets[i].x;
        offsetsMotion[i].y = offsets[i].y;
    }

    // Allocate device memory
    hipMalloc<float>(&f_kernel, kernelMotion.size());
    hipMalloc<int2>(&i_offsetsMotion, offsets.size());

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    hipMemcpy(f_kernel, kernel, kernelMotion.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(i_offsetsMotion, offsetsMotion, offsets.size() * sizeof(int2), hipMemcpyHostToDevice);

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    motionBlur << <grid, block >> > (d_input, d_output, output.cols, output.rows, f_kernel, i_offsetsMotion, kernelSize);

    hipFree(f_kernel);
    hipFree(offsetsMotion);

    delete[] i_offsetsMotion;
    delete[] f_kernel;
}











