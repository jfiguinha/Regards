#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//

#include "pictureFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include <opencv2/core/utility.hpp>
#include <opencv2/core/cuda/common.hpp>
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

 __host__ __device__ struct ColorRef
{
     uchar red;
     uchar green;
     uchar blue;
     uchar alpha;
};

inline  __host__ __device__  uchar4 GetColorSrc(unsigned char* input, int position)
{
    uchar4 value;
    value.x = input[position];
    value.y = input[position + 1];
    value.z = input[position + 2];
    value.w = input[position + 3];
    return value;
}

inline  __host__ __device__  float4 GetfColorSrc(unsigned char* input, int position)
{
    float4 value;
    value.x = input[position];
    value.y = input[position + 1];
    value.z = input[position + 2];
    value.w = input[position + 3];
    return value;
}

//----------------------------------------------------
// Conversion du NV12 vers du 32 bits
//----------------------------------------------------
inline  __host__ __device__ void rgbaFloat4ToUchar4(uchar * output, int position, float4 rgba, float fScale)
{
    output[position] = (uchar)(rgba.x * fScale);
    output[position + 1] = (uchar)(rgba.y * fScale);
    output[position + 2] = (uchar)(rgba.z * fScale);
    output[position + 3] = (uchar)(rgba.w * fScale);
    output[position] = clamp(output[position], 0, 255);
    output[position + 1] = clamp(output[position + 1], 0, 255);
    output[position + 2] = clamp(output[position + 2], 0, 255);
    output[position + 3] = clamp(output[position + 3], 0, 255);
}


inline  __host__ __device__  float4 GetColorSrc(int x, int y, uchar * input, int width, int height)
{
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        int position = (x + y * width) * 4;
        float4 value;
        value.x = (float)input[position];
        value.y = (float)input[position + 1];
        value.z = (float)input[position + 2];
        value.w = (float)input[position + 3];
        return value;
    }
    return make_float4(0.0f, 0.0f, 0.0f,0.0f);
}


//----------------------------------------------------
//Filtre Niveau de gris
//----------------------------------------------------


// Run Sharpening Filter on GPU
__global__ void solarizationFilter(uchar*srcImage, uchar*dstImage, unsigned int width, unsigned int height, int threshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float4 colorInput = GetColorSrc(x,y,srcImage,width,height);
	    float4 colorOutput = colorInput;
	    
	    float red = colorInput.x;
	    float green = colorInput.y;
	    float blue = colorInput.z;
	    float fthreshold = (float)threshold;
	    
	    if (red > fthreshold)
		    colorOutput.x = 255.0f - red;
	    else
		    colorOutput.x = red;
		    
	    if (green > fthreshold)
		    colorOutput.y = 255.0f - green;
	    else
		    colorOutput.y = green;

	    if (blue > fthreshold)
		    colorOutput.z = 255.0f - blue;
	    else
		    colorOutput.z = blue;

        dstImage[position] = colorOutput.x;
        dstImage[position + 1] = colorOutput.y;
        dstImage[position + 2] = colorOutput.z;
        dstImage[position + 3] = colorOutput.w;
    }
}


// Run Sharpening Filter on GPU
__global__ void posterisationFilter(uchar* srcImage, uchar* dstImage, unsigned int width, unsigned int height, int level)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = (x + y * width) * 4;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    uchar4 colorInput = GetColorSrc(srcImage,position);
	    uchar4 colorOutput = colorInput;
	    int _levels = max(2, min(16, level));
	    float _offset = (float)256 / (float)_levels;
	    
	    int red = colorInput.x / _offset;
	    int green = colorInput.y / _offset;
	    int blue = colorInput.z / _offset;
	    
	    colorOutput.x = (red * _offset);
	    colorOutput.y = (green * _offset);
	    colorOutput.z = (blue * _offset);

        dstImage[position] = colorOutput.x;
        dstImage[position+1] = colorOutput.y;
        dstImage[position+2] = colorOutput.z;
        dstImage[position+3] = colorOutput.w;
    }
}



//----------------------------------------------------
//Filtre Posterization
//----------------------------------------------------
__global__ void distorsionFilter(uchar* input, uchar* output,  int width, int height, float correctionRadius)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = (x + y * width) * 4;

    if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float theta = 1;
	    int halfWidth = (width / 2); 
	    int halfHeight = (height / 2);
	    
	    float newX = x - halfWidth; 
	    float newY = y - halfHeight;
	    float value = pow((float)newX,(float)2.0) + pow((float)newY, (float)2.0);
	    float distance = sqrt(value);
	    float r = distance / correctionRadius;
	    if (r != 0.0)
		    theta = atan(r) / r;
		    
	    int sourceX = round(halfWidth + theta*newX);
	    int sourceY = round(halfHeight + theta * newY);
	    
	    int positionSrc = (sourceX + sourceY * width) * 4;

	    output[position] = input[positionSrc];
        output[position+1] = input[positionSrc + 1];
        output[position + 2] = input[positionSrc + 2];
        output[position + 3] = input[positionSrc + 3];
    }
}

//---------------------------------------------------------------------
//Limite les valeurs entre 0 et 1.0f
//---------------------------------------------------------------------
inline  __host__ __device__ float4 NormalizeValue(float4 sum)
{
    float4 value;
    value.x = max(min(sum.x, 255.0f), 0.0f);
    value.y = max(min(sum.y, 255.0f), 0.0f);
    value.z = max(min(sum.z, 255.0f), 0.0f);
    value.w = max(min(sum.w, 255.0f), 0.0f);
    return value;
}

//---------------------------------------------------------------------
//Application du filtre Motion Blur
//---------------------------------------------------------------------
__global__ void motionBlur(uchar* input, uchar * output,  int width, int height, const float* kernelMotion, const int2 * offsets, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 sum = make_float4(0);
        for (int i = 0; i < kernelSize; i++)
        {
            int u = x + offsets[i].x;
            int v = y + offsets[i].y;
            if ((u < 0) || (u >= width) || (v < 0) || (v >= height))
                continue;

            float4 color = kernelMotion[i] * GetColorSrc(u, v, input, width, height);
            sum = sum + color;
        }

        rgbaFloat4ToUchar4(output, position, NormalizeValue(sum), 1.0f);

    }
}

//---------------------------------------------------------------------
//Application du filtre Soften
//	kernel = {  1, 1, 1, 1, 8, 1, 1, 1, 1 };
//	factor = 16;
//---------------------------------------------------------------------
__global__ void softenFilter(uchar* input, uchar * output,  int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 sum = GetColorSrc(x - 1, y - 1, input, width, height);
        sum += GetColorSrc(x, y - 1, input, width, height);
        sum += GetColorSrc(x + 1, y - 1, input, width, height);
        sum += GetColorSrc(x - 1, y, input, width, height);
        sum += GetColorSrc(x, y, input, width, height) * make_float4(8.0f);
        sum += GetColorSrc(x + 1, y, input, width, height);
        sum += GetColorSrc(x - 1, y + 1, input, width, height);
        sum += GetColorSrc(x, y + 1, input, width, height);
        sum += GetColorSrc(x + 1, y + 1, input, width, height);
        sum = sum / make_float4(16.0f);
        rgbaFloat4ToUchar4(output, position, sum, 1.0f);
    }
}




inline __host__ __device__ float4 make_float4(uchar4 value)
{
    float4 color = make_float4(0.0);
    color.x = value.x;
    color.y = value.y;
    color.z = value.z;
    color.w = value.w;
    return color;
}

inline __host__ __device__ uchar4 make_uchar4(float4 value)
{
    uchar4 color;;
    color.x = clamp((uchar)value.x,0,255);
    color.y = clamp((uchar)value.y, 0, 255);
    color.z = clamp((uchar)value.z, 0, 255);
    color.w = clamp((uchar)value.w, 0, 255);
    return color;
}

__global__ void cuda_filter2d(uchar* input, uchar* output, float* kernelMotion, int width, int height, int kernelSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 data = GetColorSrc(x, y, input, width, height);
        float4 sum = make_float4(0);
        for (int i = 0; i < kernelSize; i++)
        {
            float4 color = kernelMotion[i] * data;
            sum = sum + color;
        }
        rgbaFloat4ToUchar4(output, position, NormalizeValue(sum), 1.0f);
    }
}

inline  __host__ __device__ float Noise2d(int x, int y)
{
    int n = ((x + (y << 6)) << 13) ^ (x + (y << 6));
    return 0.2f * (1.0f - ((n * (n * n * 15731 + 789221) + 1376312589) & 0x7fffffff) / 1073741824.0);
}

inline  __host__ __device__ float CalculPosValue(int Xint, int Yint)
{
    int m = Xint + ((Yint) << 6);
    int n = (m << 13) ^ (m);
    return (1.0f - ((n * (n * n * 15731 + 789221) + 1376312589) & 0x7fffffff) / 1073741824.0);
}

inline  __host__ __device__ float GetValue(float x, float y)
{
    int Xint = (int)x;
    int Yint = (int)y;

    float Xfrac = x - (float)Xint;
    float Yfrac = y - (float)Yint;

    float x0y0, x1y0, x0y1, x1y1;

    if (Xint != 0 || Yint != 0)
    {
        x0y0 = CalculPosValue(Xint, Yint);
        x1y0 = CalculPosValue(Xint + 1, Yint);
        x0y1 = CalculPosValue(Xint, Yint + 1);
        x1y1 = CalculPosValue(Xint + 1, Yint + 1);
    }
    else
    {
        x0y0 = Noise2d(0, 0);
        x1y0 = Noise2d(1, 0);
        x0y1 = Noise2d(0, 1);
        x1y1 = Noise2d(1, 1);
    }

    //interpolate between those values according to the x and y fractions
    float v1 = (x0y0 + (Xfrac * (x1y0 - x0y0)));
    float v2 = (x0y1 + (Xfrac * (x1y1 - x0y1)));
    float fin = (v1 + (Yfrac * (v2 - v1)));

    return fin;
}

__global__ void noiseFilter(uchar* input, uchar * output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 n = make_float4(Noise2d(x, y));
        float4 src_color = GetColorSrc(x, y, input, width, height) + n * 255.0f;
        float4 minimal = make_float4(0.0f);
        float4 maximal = make_float4(255.0f);
        src_color = clamp(src_color, minimal, maximal);
        rgbaFloat4ToUchar4(output, position, src_color, 1.0f);
    }
}


inline  __host__ __device__ float EuclideanDist(float tcX, float tcY, float centerX, float centerY)
{
    float diffX = tcX - centerX;
    float diffY = tcY - centerY;
    return sqrt(diffX * diffX + diffY * diffY);
}

inline  __host__ __device__ float DotProduct(float tcX, float tcY, float qX, float qY)
{
    return tcX * qX + tcY * qY;
}

inline  __host__ __device__ float4 PostFX(uchar * input, int x, int y, float radius, float angleDegree, int widthIn, int heightIn)
{
    float xOut = x;
    float yOut = y;

    //Calcul du centre
    float centerX = (float)widthIn / 2.0f;
    float centerY = (float)heightIn / 2.0f;

    float tcX = (float)x - centerX;
    float tcY = (float)y - centerY;

    float angle = angleDegree * 0.0174532925;

    float dist = EuclideanDist(x, y, centerX, centerY);

    if (dist < radius)
    {
        float percent = (radius - dist) / radius;
        float theta = percent * percent * angle * 8.0;
        float s = sin(theta);
        float c = cos(theta);
        xOut = DotProduct(tcX, tcY, c, -s);
        yOut = DotProduct(tcX, tcY, s, c);

        tcX = xOut;
        tcY = yOut;
    }

    tcX += centerX;
    tcY += centerY;

    return GetColorSrc((int)tcX, (int)tcY, input, widthIn, heightIn);
}

__global__ void swirlFilter(uchar* input, uchar * output,  int width, int height, float radius, float angleDegree)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        float4 color = PostFX(input, x, y, radius, angleDegree, width, height);
        rgbaFloat4ToUchar4(output, position, color, 1.0f);
    }



inline  __host__ __device__ ColorRef GetColor(uchar* input, const int color_tid)
{
    ColorRef color;
    color.blue = input[color_tid];
    color.green = input[color_tid + 1];
    color.red = input[color_tid + 2];
    color.alpha = input[color_tid + 3];
    return color;
}

/**
 * @brief      BGR to Gray Kernel
 *
 *             This is a simple image processing kernel that converts color
 *             images to black and white by iterating over the individual
 *             pixels.
 *
 * @param      input           The input
 * @param      output          The output
 * @param[in]  width           The width
 * @param[in]  height          The height
 * @param[in]  colorWidthStep  The color width step
 * @param[in]  grayWidthStep   The gray width step
 */
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep) {
    //2D Index of current thread
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if ((xIndex < width) && (yIndex < height))
    {
        //Location of colored pixel in input
        const int color_tid = yIndex * colorWidthStep + (4 * xIndex);

        //Location of gray pixel in output
        const int gray_tid = yIndex * grayWidthStep + (4 * xIndex);

        const unsigned char blue = input[color_tid];
        const unsigned char green = input[color_tid + 1];
        const unsigned char red = input[color_tid + 2];

        const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

        output[gray_tid] = static_cast<unsigned char>(gray);
        output[gray_tid + 1] = static_cast<unsigned char>(gray);
        output[gray_tid + 2] = static_cast<unsigned char>(gray);
        output[gray_tid + 3] = static_cast<unsigned char>(gray);
    }
}


void convert_to_gray(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output) {
    // Calculate total number of bytes of input and output image
    const int colorBytes = input.step * input.rows;
    const int grayBytes = output.step * output.rows;

    unsigned char* d_input, * d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify a reasonable block size
    const dim3 block(16, 16);

    // Calculate grid size to cover the whole image
    const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);

    // Launch the color conversion kernel
    bgr_to_gray_kernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step, output.step);

    // Synchronize to check for any kernel launch errors
    cudaSafeCall(hipDeviceSynchronize(), "Kernel Launch Failed");
}


void cuda_filter2d(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const vector<float>& kernelMotion, int kernelSize)
{
    uchar* d_input;
    uchar* d_output;
    float* f_kernel;

    float* kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    hipMalloc<float>(&f_kernel, kernelMotion.size());

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    cuda_filter2d << <grid, block >> > (d_input, d_output, f_kernel, output.cols, output.rows, kernelSize);

    hipFree(f_kernel);
    delete[] kernel;
}



// The wrapper is used to call sharpening filter 
void noiseFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    noiseFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}


// The wrapper is used to call sharpening filter 
void swirlFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, float radius, float angleDegree)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    swirlFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, radius, angleDegree);
}


// The wrapper is used to call sharpening filter 
void softenFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    softenFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows);
}

// The wrapper is used to call sharpening filter 
void distorsionFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, float correctionRadius)
{
    uchar * d_input;
    uchar * d_output;

    d_input = (uchar * )input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

    // Run BoxFilter kernel on CUDA 
    distorsionFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, correctionRadius);
}



// The wrapper is used to call sharpening filter 
void posterisationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int threshold)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    posterisationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, threshold);
}


void motionBlur(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const vector<double>& kernelMotion, const vector<wxPoint>& offsets, int kernelSize)
{
    uchar* d_input;
    uchar* d_output;
    float* f_kernel;
    int2 * i_offsetsMotion;

    float * kernel = new float[kernelMotion.size()];
    for (auto i = 0; i < kernelMotion.size(); i++)
        kernel[i] = kernelMotion[i];

    int2* offsetsMotion = new int2[offsets.size()];
    for (auto i = 0; i < offsets.size(); i++)
    {
        offsetsMotion[i].x = offsets[i].x;
        offsetsMotion[i].y = offsets[i].y;
    }

    // Allocate device memory
    hipMalloc<float>(&f_kernel, kernelMotion.size());
    hipMalloc<int2>(&i_offsetsMotion, offsets.size());

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    hipMemcpy(f_kernel, kernel, kernelMotion.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(i_offsetsMotion, offsetsMotion, offsets.size() * sizeof(int2), hipMemcpyHostToDevice);

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    motionBlur << <grid, block >> > (d_input, d_output, output.cols, output.rows, f_kernel, i_offsetsMotion, kernelSize);

    hipFree(f_kernel);
    hipFree(offsetsMotion);

    delete[] i_offsetsMotion;
    delete[] f_kernel;
}


// The wrapper is used to call sharpening filter 
void solarizationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int level)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    solarizationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, level);
}













