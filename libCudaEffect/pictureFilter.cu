#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//
#include "pictureFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

// Inline device function to convert 32-bit unsigned integer to floating point rgba color 
//*****************************************************************
inline  __host__ __device__ float4 rgbaUintToFloat4(uint c)
{
    float4 rgba;
    rgba.x = c & 0xff;
    rgba.y = (c >> 8) & 0xff;
    rgba.z = (c >> 16) & 0xff;
    rgba.w = (c >> 24) & 0xff;
    return rgba;
}

inline  __host__ __device__ uchar4 rgbaUintToUChar4(uint c)
{
    uchar4 rgba;
    rgba.x = c & 0xff;
    rgba.y = (c >> 8) & 0xff;
    rgba.z = (c >> 16) & 0xff;
    rgba.w = (c >> 24) & 0xff;
    return rgba;
}

// Inline device function to convert floating point rgba color to 32-bit unsigned integer
//*****************************************************************
inline  __host__ __device__ uint rgbaFloat4ToUint(float4 rgba, float fScale)
{
    unsigned int uiPackedPix = 0U;
    uiPackedPix |= 0x000000FF & (unsigned int)(rgba.x * fScale);
    uiPackedPix |= 0x0000FF00 & (((unsigned int)(rgba.y * fScale)) << 8);
    uiPackedPix |= 0x00FF0000 & (((unsigned int)(rgba.z * fScale)) << 16);
    uiPackedPix |= 0xFF000000 & (((unsigned int)(rgba.w * fScale)) << 24);
    return uiPackedPix;
}


// Inline device function to convert floating point rgba color to 32-bit unsigned integer
//*****************************************************************
inline  __host__ __device__  uint rgbaUChar4ToUint(uchar4 rgba)
{
    unsigned int uiPackedPix = 0U;
    uiPackedPix |= 0x000000FF & (unsigned int)(rgba.x);
    uiPackedPix |= 0x0000FF00 & (((unsigned int)(rgba.y)) << 8);
    uiPackedPix |= 0x00FF0000 & (((unsigned int)(rgba.z)) << 16);
    uiPackedPix |= 0xFF000000 & (((unsigned int)(rgba.w)) << 24);
    return uiPackedPix;
}


//----------------------------------------------------
//Filtre Niveau de gris
//----------------------------------------------------

//---------------------------------------------------------------------
//Limite les valeurs entre 0 et 1.0f
//---------------------------------------------------------------------
inline  __host__ __device__ float4 NormalizeValue(float4 sum)
{
	float4 value;
	value.x = max(min(sum.x, 255.0f), 0.0f);
	value.y = max(min(sum.y, 255.0f), 0.0f);
	value.z = max(min(sum.z, 255.0f), 0.0f);  
	value.w = max(min(sum.w, 255.0f), 0.0f); 
	return value;
}


// Run Sharpening Filter on GPU
__global__ void mosaicFilter(uint *srcImage, uint *dstImage, unsigned int width, unsigned int height, int fTileSize)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float w = fTileSize;
	    float h = fTileSize;		

	    float s = floor(x / w);
	    float t = floor(y / h);	
	    
	    int sourcex = __float2int_rd(s * w);
	    int sourcey = __float2int_rd(t * h);
	    
	    int positionSrc = sourcex + sourcey * width;
	    int positionDest = x + y * width;

	    dstImage[positionDest] = srcImage[positionSrc];
    }
}


// Run Sharpening Filter on GPU
__global__ void solarizationFilter(uint *srcImage, uint *dstImage, unsigned int width, unsigned int height, int threshold)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   
   int position = x + y * width;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float4 colorInput = rgbaUintToFloat4(srcImage[position]);
	    float4 colorOutput = colorInput;
	    
	    float red = colorInput.x;
	    float green = colorInput.y;
	    float blue = colorInput.z;
	    float fthreshold = (float)threshold;
	    
	    if (red > fthreshold)
		    colorOutput.x = 255.0f - red;
	    else
		    colorOutput.x = red;
		    
	    if (green > fthreshold)
		    colorOutput.y = 255.0f - green;
	    else
		    colorOutput.y = green;

	    if (blue > fthreshold)
		    colorOutput.z = 255.0f - blue;
	    else
		    colorOutput.z = blue;

	    dstImage[position] = rgbaFloat4ToUint(colorOutput, 1.0f);
    }
}


// Run Sharpening Filter on GPU
__global__ void posterisationFilter(uint * srcImage, uint * dstImage, unsigned int width, unsigned int height, int level)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = x + y * width;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    uchar4 colorInput = rgbaUintToUChar4(srcImage[position]);
	    uchar4 colorOutput = colorInput;
	    int _levels = max(2, min(16, level));
	    float _offset = (float)256 / (float)_levels;
	    
	    int red = colorInput.x / _offset;
	    int green = colorInput.y / _offset;
	    int blue = colorInput.z / _offset;
	    
	    colorOutput.x = (red * _offset);
	    colorOutput.y = (green * _offset);
	    colorOutput.z = (blue * _offset);

        dstImage[position] = rgbaUChar4ToUint(colorOutput);
    }
}

//----------------------------------------------------
//Filtre Posterization
//----------------------------------------------------
__global__ void distorsionFilter(uint *output, uint * input, int width, int height, float correctionRadius)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = x + y * width;

    if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float theta = 1;
	    int halfWidth = (width / 2); 
	    int halfHeight = (height / 2);
	    
	    float newX = x - halfWidth; 
	    float newY = y - halfHeight;
	    float value = pow((float)newX,(float)2.0) + pow((float)newY, (float)2.0);
	    float distance = sqrt(value);
	    float r = distance / correctionRadius;
	    if (r != 0.0)
		    theta = atan(r) / r;
		    
	    int sourceX = round(halfWidth + theta*newX);
	    int sourceY = round(halfHeight + theta * newY);
	    
	    int positionSrc = sourceX + sourceY * width;
	    int position = x + y * width;

	    output[position] = input[positionSrc];
    }
}


// The wrapper is used to call sharpening filter 
void distorsionFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, float correctionRadius)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = 4;//input.step/input.cols; 

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows * channel;
        const int outputSize = output.cols * output.rows * channel;
        uint *d_input, *d_output;
        
        // Allocate device memory
        hipMalloc<uint>(&d_input,inputSize);
        hipMalloc<uint>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyDeviceToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        distorsionFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, correctionRadius);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToDevice);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}



// The wrapper is used to call sharpening filter 
void posterisationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int threshold)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = 4;//input.step/input.cols; 

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows * channel;
        const int outputSize = output.cols * output.rows * channel;
        uint *d_input, *d_output;
        
        // Allocate device memory
        hipMalloc<uint>(&d_input,inputSize);
        hipMalloc<uint>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyDeviceToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        posterisationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, threshold);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToDevice);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}



// The wrapper is used to call sharpening filter 
void solarizationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int level)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = 4;//input.step/input.cols; 

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows * channel;
        const int outputSize = output.cols * output.rows * channel;
        uint *d_input, *d_output;
        
        // Allocate device memory
        hipMalloc<uint>(&d_input,inputSize);
        hipMalloc<uint>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyDeviceToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        solarizationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, level);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToDevice);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}


// The wrapper is used to call sharpening filter 
void mosaicFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int fTileSize)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = 4;//input.step/input.cols; 

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows * channel;
        const int outputSize = output.cols * output.rows * channel;
        uint *d_input, *d_output;
        
        // Allocate device memory
        hipMalloc<uint>(&d_input,inputSize);
        hipMalloc<uint>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyDeviceToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        mosaicFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, fTileSize);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToDevice);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}










