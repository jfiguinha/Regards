#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//
#include "pictureFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

inline  __host__ __device__  uchar4 GetColorSrc(unsigned char* input, int position)
{
    uchar4 value;
    value.x = input[position];
    value.y = input[position + 1];
    value.z = input[position + 2];
    value.w = input[position + 3];
    return value;
}

//----------------------------------------------------
// Conversion du NV12 vers du 32 bits
//----------------------------------------------------
inline  __host__ __device__ void rgbaFloat4ToUchar4(unsigned char*& output, int position, float4 rgba, float fScale)
{
    output[position] = (uchar)(rgba.x * fScale);
    output[position + 1] = (uchar)(rgba.y * fScale);
    output[position + 2] = (uchar)(rgba.z * fScale);
    output[position + 3] = (uchar)(rgba.w * fScale);

}


inline  __host__ __device__  float4 GetColorSrc(int x, int y, unsigned char* input, int width, int height)
{
    if (x < width && y < height && y >= 0 && x >= 0)
    {
        int position = (x + y * width) * 4;
        float4 value;
        value.x = (float)input[position];
        value.y = (float)input[position + 1];
        value.z = (float)input[position + 2];
        value.w = (float)input[position + 3];
        return value;
    }
    return make_float4(0.0f, 0.0f, 0.0f,0.0f);
}


//----------------------------------------------------
//Filtre Niveau de gris
//----------------------------------------------------


// Run Sharpening Filter on GPU
__global__ void mosaicFilter(uchar*srcImage, uchar*dstImage, unsigned int width, unsigned int height, int fTileSize)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float w = fTileSize;
	    float h = fTileSize;		

	    float s = floor(x / w);
	    float t = floor(y / h);	
	    
	    int sourcex = __float2int_rd(s * w);
	    int sourcey = __float2int_rd(t * h);
	    
	    int positionSrc = (sourcex + sourcey * width) * 4;
	    int positionDest = (x + y * width) * 4;

        dstImage[positionDest] = srcImage[positionSrc];
        dstImage[positionDest + 1] = srcImage[positionSrc + 1];
        dstImage[positionDest + 2] = srcImage[positionSrc + 2];
        dstImage[positionDest + 3] = srcImage[positionSrc + 3];
    }
}


// Run Sharpening Filter on GPU
__global__ void solarizationFilter(uchar*srcImage, uchar*dstImage, unsigned int width, unsigned int height, int threshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float4 colorInput = GetColorSrc(x,y,srcImage,width,height);
	    float4 colorOutput = colorInput;
	    
	    float red = colorInput.x;
	    float green = colorInput.y;
	    float blue = colorInput.z;
	    float fthreshold = (float)threshold;
	    
	    if (red > fthreshold)
		    colorOutput.x = 255.0f - red;
	    else
		    colorOutput.x = red;
		    
	    if (green > fthreshold)
		    colorOutput.y = 255.0f - green;
	    else
		    colorOutput.y = green;

	    if (blue > fthreshold)
		    colorOutput.z = 255.0f - blue;
	    else
		    colorOutput.z = blue;

        dstImage[position] = colorOutput.x;
        dstImage[position + 1] = colorOutput.y;
        dstImage[position + 2] = colorOutput.z;
        dstImage[position + 3] = colorOutput.w;
    }
}


// Run Sharpening Filter on GPU
__global__ void posterisationFilter(uchar* srcImage, uchar* dstImage, unsigned int width, unsigned int height, int level)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = (x + y * width) * 4;

	if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    uchar4 colorInput = GetColorSrc(srcImage,position);
	    uchar4 colorOutput = colorInput;
	    int _levels = max(2, min(16, level));
	    float _offset = (float)256 / (float)_levels;
	    
	    int red = colorInput.x / _offset;
	    int green = colorInput.y / _offset;
	    int blue = colorInput.z / _offset;
	    
	    colorOutput.x = (red * _offset);
	    colorOutput.y = (green * _offset);
	    colorOutput.z = (blue * _offset);

        dstImage[position] = colorOutput.x;
        dstImage[position+1] = colorOutput.y;
        dstImage[position+2] = colorOutput.z;
        dstImage[position+3] = colorOutput.w;
    }
}



//----------------------------------------------------
//Filtre Posterization
//----------------------------------------------------
__global__ void distorsionFilter(uchar* output, uchar* input, int width, int height, float correctionRadius)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = (x + y * width) * 4;

    if(x < width && y < height && y >= 0 && x >= 0)	
    {
	    float theta = 1;
	    int halfWidth = (width / 2); 
	    int halfHeight = (height / 2);
	    
	    float newX = x - halfWidth; 
	    float newY = y - halfHeight;
	    float value = pow((float)newX,(float)2.0) + pow((float)newY, (float)2.0);
	    float distance = sqrt(value);
	    float r = distance / correctionRadius;
	    if (r != 0.0)
		    theta = atan(r) / r;
		    
	    int sourceX = round(halfWidth + theta*newX);
	    int sourceY = round(halfHeight + theta * newY);
	    
	    int positionSrc = (sourceX + sourceY * width) * 4;

	    output[position] = input[positionSrc];
        output[position+1] = input[positionSrc + 1];
        output[position + 2] = input[positionSrc + 2];
        output[position + 3] = input[positionSrc + 3];
    }
}

__global__ void sharpenMasking(uchar* output, uchar * input, uchar * gaussian, int width, int height, float sharpness)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int position = (x + y * width) * 4;
    if (x < width && y < height && y >= 0 && x >= 0)
    {

        float4 origin = GetColorSrc(x, y, input, width, height);
        float4 color = GetColorSrc(x, y, gaussian, width, height);

        color = origin - color;

        float4 value = origin + color * sharpness;


        value = clamp(value, 255.0f, 0.0f);

        rgbaFloat4ToUchar4(output, position, value, 1.0f);
    }
}


// The wrapper is used to call sharpening filter 
void distorsionFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, float correctionRadius)
{
    uchar * d_input;
    uchar * d_output;

    d_input = (uchar * )input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

    // Run BoxFilter kernel on CUDA 
    distorsionFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, correctionRadius);
}



// The wrapper is used to call sharpening filter 
void posterisationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int threshold)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    posterisationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, threshold);
}



// The wrapper is used to call sharpening filter 
void solarizationFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int level)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    solarizationFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, level);
}


// The wrapper is used to call sharpening filter 
void mosaicFilter(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, int fTileSize)
{
    uchar* d_input;
    uchar* d_output;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    mosaicFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, fTileSize);

}

// The wrapper is used to call sharpening filter 
void sharpenMasking(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output, const cv::cuda::GpuMat& gaussian, float sharpness)
{
    uchar* d_input;
    uchar* d_output;
    uchar* gaussian_input;

    d_input = (uchar*)input.ptr();
    d_output = (uchar*)output.ptr();
    gaussian_input = (uchar*)gaussian.ptr();

    // Specify block size
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // Calculate grid size to cover the whole image
    const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

    // Run BoxFilter kernel on CUDA 
    sharpenMasking << <grid, block >> > (d_input, d_output, gaussian_input, output.cols, output.rows, sharpness);

}










