#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//
#include "videoFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

//----------------------------------------------------
// Conversion du NV12 vers du 32 bits
//----------------------------------------------------
inline  __host__ __device__ uint rgbaFloat4ToUint(float4 rgba, float fScale)
{
    unsigned int uiPackedPix = 0U;
    uiPackedPix |= 0x000000FF & (unsigned int)(rgba.x * fScale);
    uiPackedPix |= 0x0000FF00 & (((unsigned int)(rgba.y * fScale)) << 8);
    uiPackedPix |= 0x00FF0000 & (((unsigned int)(rgba.z * fScale)) << 16);
    uiPackedPix |= 0xFF000000 & (((unsigned int)(rgba.w * fScale)) << 24);
    return uiPackedPix;
}



//----------------------------------------------------
// Conversion Special Effect Video du NV12 vers le RGB32
//----------------------------------------------------
__global__ void convertYUVtoRGB32(uint *output, const uchar *inputY, const uchar *inputU, const uchar *inputV, int widthIn, int heightIn, int widthOut, int heightOut, int pitch, int colorRange, int colorSpace) 
{ 
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = x + y * widthOut;
	if(x < widthOut && y < heightOut && y >= 0 && x >= 0)	
	{
		float4 color;
		int positionSrc = x + y * pitch;
		int positionUV = 0;
		if (x & 1)
		{		
			if (y & 1)
				positionUV = ((x - 1) / 2) + ((y - 1) / 2) * (pitch / 2);
			else
				positionUV = ((x - 1) / 2) + (y / 2) * (pitch / 2);
		}
		else
		{
			if (y & 1)
				positionUV = (x / 2) + ((y - 1) / 2) * (pitch / 2);
			else
				positionUV = (x / 2) + (y / 2) * (pitch / 2);
		}
		float uComp = inputU[positionUV];
		float vComp = inputV[positionUV];
		float yComp = inputY[positionSrc];
		    // RGB conversion
		
		float3 matrix[3];
		if(colorSpace == 0)
		{
			//default
			matrix[0].x = 1.164;
			matrix[0].y = 0;
			matrix[0].z = 1.596;
			
			matrix[1].x = 1.164;
			matrix[1].y = -0.391;
			matrix[1].z = -0.813;

			matrix[2].x = 1.164;
			matrix[2].y = 2.018;
			matrix[2].z = 0;			
		}
		else if(colorSpace == 1)
		{
			//bt601
			matrix[0].x = 1;
			matrix[0].y = 0;
			matrix[0].z = 1;
			
			matrix[1].x = 1;
			matrix[1].y = -0.344;
			matrix[1].z = -0.714;

			matrix[2].x = 1;
			matrix[2].y = 1.772;
			matrix[2].z = 0;
		}
		else if(colorSpace == 2)
		{
			//BT.709
			matrix[0].x = 1;
			matrix[0].y = 0;
			matrix[0].z = 1.5748;
			
			matrix[1].x = 1;
			matrix[1].y = -0.187324;
			matrix[1].z = -0.468124;

			matrix[2].x = 1;
			matrix[2].y = 1.8556;
			matrix[2].z = 0;
		}		
		else if(colorSpace == 3)
		{
			//BT.2020
			matrix[0].x = 1;
			matrix[0].y = 0;
			matrix[0].z = 1.402;
			
			matrix[1].x = 1;
			matrix[1].y = -0.344136286;
			matrix[1].z = -0.7141362862;

			matrix[2].x = 1;
			matrix[2].y = 1.772;
			matrix[2].z = 0;
		}	
		
		color.z = (matrix[0].x * (yComp - 16) + matrix[0].y * (uComp-128) + matrix[0].z * (vComp-128));
		color.y = (matrix[1].x * (yComp - 16) + matrix[1].y * (uComp-128) + matrix[1].z * (vComp-128));
		color.x = (matrix[2].x * (yComp - 16) + matrix[2].y * (uComp-128) + matrix[2].z * (vComp-128));
		color.w = 255.0f;

		float4 minimal = make_float4(0.0);
		float4 maximal = make_float4(255.0);
		
		if(colorRange == 1)
		{
			minimal = make_float4(16.0);
			maximal = make_float4(235.0);
		}
        
		

		color = clamp(color,minimal,maximal);
		output[position] = rgbaFloat4ToUint(color,1.0f);
	}
} 


//----------------------------------------------------
// Conversion Special Effect Video du NV12 vers le RGB32
//----------------------------------------------------
__global__ void convertNV12toRGB32(uint * output,  unsigned char * inputY, unsigned char * inputUV, int widthIn, int heightIn, int widthOut, int heightOut, int pitch, int colorRange, int colorSpace) 
{ 
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int position = x + y * widthOut;
	if(x < widthOut && y < heightOut && y >= 0 && x >= 0)	
	{
		//float4 color = getColorFromNV12(inputY, inputUV, x,  y, widthIn, heightIn, pitch, colorRange, colorSpace); 
        
		float4 color;
		int positionSrc = x + y * pitch;
		int positionUV = 0;

		int yModulo = y % 2;
		int xModulo = x % 2;
		if (xModulo == 1)
		{		
			if (yModulo == 1)
				positionUV = (x - 1) + ((y - 1) / 2) * pitch;
			else
				positionUV = (x - 1) + (y / 2) * pitch;
		}
		else
		{
			if (yModulo == 1)
				positionUV = x + ((y - 1) / 2) * pitch;
			else
				positionUV = x + (y / 2) * pitch;
		}
		
		uchar vComp = inputUV[positionUV];
		uchar uComp = inputUV[positionUV + 1];
		uchar yComp = inputY[positionSrc];
		
		float3 matrix[3];
		if(colorSpace == 0)
		{
			//default
			matrix[0].x = 1.164;
			matrix[0].y = 0;
			matrix[0].z = 1.596;
			
			matrix[1].x = 1.164;
			matrix[1].y = -0.391;
			matrix[1].z = -0.813;

			matrix[2].x = 1.164;
			matrix[2].y = 2.018;
			matrix[2].z = 0;			
		}
		else if(colorSpace == 1)
		{
			//bt601
			matrix[0].x = 1;
			matrix[0].y = 0;
			matrix[0].z = 1;
			
			matrix[1].x = 1;
			matrix[1].y = -0.344;
			matrix[1].z = -0.714;

			matrix[2].x = 1;
			matrix[2].y = 1.772;
			matrix[2].z = 0;
		}
		else if(colorSpace == 2)
		{
			//BT.709
			matrix[0].x = 1;
			matrix[0].y = 0;
			matrix[0].z = 1.5748;
			
			matrix[1].x = 1;
			matrix[1].y = -0.187324;
			matrix[1].z = -0.468124;

			matrix[2].x = 1;
			matrix[2].y = 1.8556;
			matrix[2].z = 0;
		}		
		else if(colorSpace == 3)
		{
			//BT.2020
			matrix[0].x = 1;
			matrix[0].y = 0;
			matrix[0].z = 1.402;

			matrix[1].x = 1;
			matrix[1].y = -0.344136286;
			matrix[1].z = -0.7141362862;

			matrix[2].x = 1;
			matrix[2].y = 1.772;
			matrix[2].z = 0;
		}	
		
		color.x = (matrix[0].x * (yComp - 16) + matrix[0].y * (uComp-128) + matrix[0].z * (vComp-128));
		color.y = (matrix[1].x * (yComp - 16) + matrix[1].y * (uComp-128) + matrix[1].z * (vComp-128));
		color.z = (matrix[2].x * (yComp - 16) + matrix[2].y * (uComp-128) + matrix[2].z * (vComp-128));
		color.w = 255.0f;

		float4 minimal = make_float4(0.0);
		float4 maximal = make_float4(255.0);
		
		if(colorRange == 1)
		{
			minimal = make_float4(16.0);
			maximal = make_float4(235.0);
		}
        
        color = clamp(color,minimal,maximal);
        
		output[position] = rgbaFloat4ToUint(color,1.0f);
	}
} 

// The wrapper is used to call sharpening filter 
void convertNV12toRGB32(cv::cuda::GpuMat& output, const cv::Mat& inputY, const cv::Mat& inputUV, int pitch, int colorRange, int colorSpace)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = 4;//inputY.step/inputY.cols; 

        // Calculate number of input & output bytes in each block
        const int inputYSize = inputY.cols * inputY.rows;
        const int inputUVSize = inputUV.cols * inputUV.rows;
        const int outputSize = output.cols * output.rows * channel;
        unsigned char *d_inputY, * d_inputUV;
        uint * d_output;
        
        // Allocate device memory
        hipMalloc<unsigned char>(&d_inputY,inputYSize);
        hipMalloc<unsigned char>(&d_inputUV,inputUVSize);
        hipMalloc<uint>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_inputY,inputY.ptr(),inputYSize,hipMemcpyHostToDevice);
        hipMemcpy(d_inputUV,inputUV.ptr(),inputUVSize,hipMemcpyHostToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        convertNV12toRGB32<<<grid,block>>>(d_output,d_inputY,d_inputUV, inputY.cols, inputY.rows, output.cols, output.rows, pitch, colorRange, colorSpace);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToDevice);

        //Free the device memory
        hipFree(d_inputY);
        hipFree(d_inputUV);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}


// The wrapper is used to call sharpening filter 
void convertYUVtoRGB32(cv::cuda::GpuMat& output, const cv::Mat& inputY, const cv::Mat& inputU, const cv::Mat& inputV, int pitch, int colorRange, int colorSpace)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = 4;//inputY.step/inputY.cols; 

        // Calculate number of input & output bytes in each block
        const int inputYSize = inputY.cols * inputY.rows;
        const int inputUSize = inputU.cols * inputU.rows;
        const int inputVSize = inputV.cols * inputV.rows;
        const int outputSize = output.cols * output.rows * channel;
        unsigned char *d_inputY, * d_inputU, * d_inputV;
        uint * d_output;
        
        // Allocate device memory
        hipMalloc<unsigned char>(&d_inputY,inputYSize);
        hipMalloc<unsigned char>(&d_inputU,inputUSize);
        hipMalloc<unsigned char>(&d_inputV,inputVSize);
        hipMalloc<uint>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_inputY,inputY.ptr(),inputYSize,hipMemcpyHostToDevice);
        hipMemcpy(d_inputU,inputU.ptr(),inputUSize,hipMemcpyHostToDevice);
        hipMemcpy(d_inputV,inputV.ptr(),inputVSize,hipMemcpyHostToDevice);
        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        convertYUVtoRGB32<<<grid,block>>>(d_output,d_inputY,d_inputU,d_inputV, inputY.cols, inputY.rows, output.cols, output.rows, pitch, colorRange, colorSpace);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToDevice);

        //Free the device memory
        hipFree(d_inputY);
        hipFree(d_inputU);
        hipFree(d_inputV);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}