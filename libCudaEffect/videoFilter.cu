#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//
#include "videoFilter.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

//----------------------------------------------------
// Conversion du NV12 vers du 32 bits
//----------------------------------------------------
inline  __host__ __device__ void rgbaFloat4ToUchar4(unsigned char* & output, int position, float4 rgba, float fScale)
{
    output[position] = (uchar)(rgba.x * fScale);
    output[position+1] = (uchar)(rgba.y * fScale);
    output[position+2] = (uchar)(rgba.z * fScale);
    output[position+3] = (uchar)(rgba.w * fScale);

}

inline  __host__ __device__ float4 yuvToRgb32(float yComp, float uComp, float vComp, int colorRange, int colorSpace)
{
    float4 color = make_float4(0.0);
    float3 matrix[3];
    if(colorSpace == 0)
    {
        //default
        matrix[0].x = 1.164;
        matrix[0].y = 0;
        matrix[0].z = 1.596;
        
        matrix[1].x = 1.164;
        matrix[1].y = -0.391;
        matrix[1].z = -0.813;

        matrix[2].x = 1.164;
        matrix[2].y = 2.018;
        matrix[2].z = 0;			
    }
    else if(colorSpace == 1)
    {
        //bt601
        matrix[0].x = 1;
        matrix[0].y = 0;
        matrix[0].z = 1;
        
        matrix[1].x = 1;
        matrix[1].y = -0.344;
        matrix[1].z = -0.714;

        matrix[2].x = 1;
        matrix[2].y = 1.772;
        matrix[2].z = 0;
    }
    else if(colorSpace == 2)
    {
        //BT.709
        matrix[0].x = 1;
        matrix[0].y = 0;
        matrix[0].z = 1.5748;
        
        matrix[1].x = 1;
        matrix[1].y = -0.187324;
        matrix[1].z = -0.468124;

        matrix[2].x = 1;
        matrix[2].y = 1.8556;
        matrix[2].z = 0;
    }		
    else if(colorSpace == 3)
    {
        //BT.2020
        matrix[0].x = 1;
        matrix[0].y = 0;
        matrix[0].z = 1.402;
        
        matrix[1].x = 1;
        matrix[1].y = -0.344136286;
        matrix[1].z = -0.7141362862;

        matrix[2].x = 1;
        matrix[2].y = 1.772;
        matrix[2].z = 0;
    }	
    
    color.z = (matrix[0].x * (yComp - 16) + matrix[0].y * (uComp-128) + matrix[0].z * (vComp-128));
    color.y = (matrix[1].x * (yComp - 16) + matrix[1].y * (uComp-128) + matrix[1].z * (vComp-128));
    color.x = (matrix[2].x * (yComp - 16) + matrix[2].y * (uComp-128) + matrix[2].z * (vComp-128));
    color.w = 255.0f;

    float4 minimal = make_float4(0.0);
    float4 maximal = make_float4(255.0);
    
    if(colorRange == 1)
    {
        minimal = make_float4(16.0);
        maximal = make_float4(235.0);
    }
    
    

    color = clamp(color,minimal,maximal);

    return color;
}

//----------------------------------------------------
// Conversion Special Effect Video du NV12 vers le RGB32
//----------------------------------------------------
__global__ void convertYUVtoRGB32(unsigned char * output, const uchar *inputY, const uchar *inputU, const uchar *inputV, int widthIn, int heightIn, int lineIn, int widthOut, int heightOut, int pitch, int colorRange, int colorSpace)
{ 
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < widthOut && y < heightOut && y >= 0 && x >= 0)	
	{
		float4 color;
        const int position = y * pitch + (4 * x);
		int positionSrc = x + y * lineIn;
		int positionUV = 0;
		if (x & 1)
		{		
			if (y & 1)
				positionUV = ((x - 1) / 2) + ((y - 1) / 2) * (lineIn / 2);
			else
				positionUV = ((x - 1) / 2) + (y / 2) * (lineIn / 2);
		}
		else
		{
			if (y & 1)
				positionUV = (x / 2) + ((y - 1) / 2) * (lineIn / 2);
			else
				positionUV = (x / 2) + (y / 2) * (lineIn / 2);
		}
		float uComp = inputU[positionUV];
		float vComp = inputV[positionUV];
		float yComp = inputY[positionSrc];
		    // RGB conversion
		color = yuvToRgb32(yComp, uComp, vComp, colorRange, colorSpace);

        rgbaFloat4ToUchar4(output, position, color, 1.0f);
	}
} 


//----------------------------------------------------
// Conversion Special Effect Video du NV12 vers le RGB32
//----------------------------------------------------
__global__ void convertNV12toRGB32(unsigned char * output,  unsigned char * inputY, unsigned char * inputUV, int widthIn, int heightIn, int lineIn, int widthOut, int heightOut, int pitch, int colorRange, int colorSpace)
{ 
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   
	if(x < widthOut && y < heightOut && y >= 0 && x >= 0)	
	{
		//float4 color = getColorFromNV12(inputY, inputUV, x,  y, widthIn, heightIn, pitch, colorRange, colorSpace); 
        const int position = y * pitch + (4 * x);
		float4 color;
		int positionSrc = x + y * lineIn;
		int positionUV = 0;

		int yModulo = y % 2;
		int xModulo = x % 2;
		if (xModulo == 1)
		{		
			if (yModulo == 1)
				positionUV = (x - 1) + ((y - 1) / 2) * lineIn;
			else
				positionUV = (x - 1) + (y / 2) * lineIn;
		}
		else
		{
			if (yModulo == 1)
				positionUV = x + ((y - 1) / 2) * lineIn;
			else
				positionUV = x + (y / 2) * lineIn;
		}
		
		uchar vComp = inputUV[positionUV];
		uchar uComp = inputUV[positionUV + 1];
		uchar yComp = inputY[positionSrc];
        
		color = yuvToRgb32(yComp, uComp, vComp, colorRange, colorSpace);
        
        rgbaFloat4ToUchar4(output, position, color, 1.0f);

	}
} 

// The wrapper is used to call sharpening filter 
void convertNV12toRGB32(cv::cuda::GpuMat& output, uint8_t* bufferY, int sizeY, uint8_t* bufferUV, int sizeUV, const int& width,
    const int& height, const int& lineSize, const int& widthOut, const int& heightOut,
    const int& colorRange, const int& colorSpace)
{
        unsigned char *d_inputY, * d_inputUV;
        unsigned char * d_output = output.ptr();
        
        // Allocate device memory
        hipMalloc<unsigned char>(&d_inputY, sizeY);
        hipMalloc<unsigned char>(&d_inputUV, sizeUV);
        //hipMalloc<unsigned char>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_inputY, bufferY, sizeY,hipMemcpyHostToDevice);
        hipMemcpy(d_inputUV, bufferUV, sizeUV,hipMemcpyHostToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Run BoxFilter kernel on CUDA 
        convertNV12toRGB32<<<grid,block>>>(d_output,d_inputY,d_inputUV, width, height, lineSize, widthOut, heightOut, output.step, colorRange, colorSpace);

        //Free the device memory
        hipFree(d_inputY);
        hipFree(d_inputUV);
}


// The wrapper is used to call sharpening filter 
void convertYUVtoRGB32(cv::cuda::GpuMat& output, uint8_t* bufferY, int sizeY, uint8_t* bufferU, int sizeU, uint8_t* bufferV, int sizeV,
    const int& width, const int& height, const int& lineSize, const int& widthOut,
    const int& heightOut, const int& colorRange, const int& colorSpace)
{
        unsigned char *d_inputY, * d_inputU, * d_inputV;
        unsigned char* d_output = output.ptr();
        
        // Allocate device memory
        hipMalloc<unsigned char>(&d_inputY, sizeY);
        hipMalloc<unsigned char>(&d_inputU, sizeU);
        hipMalloc<unsigned char>(&d_inputV, sizeV);
       // hipMalloc<unsigned char>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_inputY, bufferY, sizeY,hipMemcpyHostToDevice);
        hipMemcpy(d_inputU, bufferU, sizeU,hipMemcpyHostToDevice);
        hipMemcpy(d_inputV, bufferV, sizeV,hipMemcpyHostToDevice);
        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Run BoxFilter kernel on CUDA 
        convertYUVtoRGB32<<<grid,block>>>(d_output,d_inputY,d_inputU,d_inputV, width, height, lineSize, widthOut, heightOut, output.step, colorRange, colorSpace);

        //Free the device memory
        hipFree(d_inputY);
        hipFree(d_inputU);
        hipFree(d_inputV);
}